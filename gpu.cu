#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <assert.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include "common.h"

#define NUM_THREADS 256
#define density 0.0005
#define num_particles_per_bin 500

extern double size;
//
//  benchmarking program
//

typedef struct
{
    int particles[num_particles_per_bin];
    int length;
} sampleBin;


__device__ void apply_force_gpu(particle_t &particle, particle_t &neighbor)
{
    double dx = neighbor.x - particle.x;
    double dy = neighbor.y - particle.y;
    double r2 = dx * dx + dy * dy;
    if( r2 > cutoff*cutoff )
        return;
    //r2 = fmax( r2, min_r*min_r );
    r2 = (r2 > min_r*min_r) ? r2 : min_r*min_r;
    double r = sqrt( r2 );

    //
    //  very simple short-range repulsive force
    //
    double coef = ( 1 - cutoff / r ) / r2 / mass;
    particle.ax += coef * dx;
    particle.ay += coef * dy;

}

__global__ void compute_forces_gpu(particle_t * particles, sampleBin* bins, int n_rows, int n_cols, double bin_side, int n)
{
    // Get thread (particle) ID
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if(tid >= n) return;
    particles[tid].ax = particles[tid].ay = 0;

    int row = floor(particles[tid].x / bin_side);
    int col = floor(particles[tid].y / bin_side);

    for (int i = -1; i < 2; i++) {
        for (int j = -1; j < 2; j++) {
            int bin_ind_x = row + i;
            int bin_ind_y = col + j;
            if (bin_ind_x >= 0 && bin_ind_x < n_rows && bin_ind_y >= 0 && bin_ind_y < n_cols) {
                int neighbor_ind = bin_ind_x*n_cols + bin_ind_y;
                sampleBin& neighbor_bin = bins[neighbor_ind];
                for (int z = 0; z<neighbor_bin.length; z++) {
                    particle_t& neighbor = particles[neighbor_bin.particles[z]];
                    apply_force_gpu(particles[tid], neighbor);
                }
            }
        }
    }
}

__global__ void move_gpu (particle_t * particles, int n, double size)
{

    // Get thread (particle) ID
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if(tid >= n) return;

    particle_t * p = &particles[tid];
    //
    //  slightly simplified Velocity Verlet integration
    //  conserves energy better than explicit Euler method
    //
    p->vx += p->ax * dt;
    p->vy += p->ay * dt;
    p->x  += p->vx * dt;
    p->y  += p->vy * dt;

    while( p->x < 0 || p->x > size )
    {
        p->x  = p->x < 0 ? -(p->x) : 2*size-p->x;
        p->vx = -(p->vx);
    }
    while( p->y < 0 || p->y > size )
    {
        p->y  = p->y < 0 ? -(p->y) : 2*size-p->y;
        p->vy = -(p->vy);
    }

}

__global__ void reassign_bins (particle_t* particles, sampleBin* bins, int roll, int num_bins, double bin_side, int n_cols) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if(tid >= num_bins) return;

    sampleBin &bin = bins[tid];
    bin.length = 0;
    for (int i = 0; i < roll; i++) {
        int x_ind = floor(particles[i].x / bin_side);
        int y_ind = floor(particles[i].y / bin_side);
        if (x_ind * n_cols + y_ind == tid) {
            bin.particles[bin.length] = i;
            bin.length += 1;
        }
    }
}


int main( int argc, char **argv )
{
    // This takes a few seconds to initialize the runtime
    hipDeviceSynchronize();

    if( find_option( argc, argv, "-h" ) >= 0 )
    {
        printf( "Options:\n" );
        printf( "-h to see this help\n" );
        printf( "-n <int> to set the number of particles\n" );
        printf( "-o <filename> to specify the output file name\n" );
        return 0;
    }

    int n = read_int( argc, argv, "-n", 1000 );

    char *savename = read_string( argc, argv, "-o", NULL );

    FILE *fsave = savename ? fopen( savename, "w" ) : NULL;
    particle_t *particles = (particle_t*) malloc( n * sizeof(particle_t) );

    // GPU particle data structure
    particle_t * d_particles;
    hipMalloc((void **) &d_particles, n * sizeof(particle_t));

    set_size( n );

    init_particles( n, particles );

    hipDeviceSynchronize();
    double copy_time = read_timer( );

    // Copy the particles to the GPU
    hipMemcpy(d_particles, particles, n * sizeof(particle_t), hipMemcpyHostToDevice);

    int n_rows = ceil(sqrt(NUM_THREADS));
    int n_cols = n_rows;
    int cross = n_rows * n_cols;
    double bin_side = size / n_rows;

    sampleBin* cpu_bins = (Bin* ) malloc(cross * sizeof(Bin));
    for (int i = 0; i < cross; i++) {
        cpu_bins[i] = sampleBin();
        cpu_bins[i].length = 0;
        for (int j = 0; j< num_particles_per_bin; j++) {
            cpu_bins[i].particles[j] = -1;
        }
    }

    for ( int i = 0; i < n; i++) {
        int x_ind = floor(particles[i].x / bin_side);
        int y_ind = floor(particles[i].y / bin_side);
        cpu_bins[x_ind*n_cols + y_ind].particles[cpu_bins[x_ind*n_cols + y_ind].length] = i;
        cpu_bins[x_ind*n_cols + y_ind].length += 1;
    }

    sampleBin* gpu_bins;
    hipMalloc((void **) &gpu_bins, cross * sizeof(sampleBin));

    hipMemcpy(gpu_bins, cpu_bins, cross * sizeof(sampleBin), hipMemcpyHostToDevice);

    hipDeviceSynchronize();
    copy_time = read_timer( ) - copy_time;

    //
    //  simulate a number of time steps
    //
    hipDeviceSynchronize();
    double simulation_time = read_timer( );
    int blocks = (n + NUM_THREADS - 1) / NUM_THREADS;
    int sqrt_num_threads = floor(sqrt(NUM_THREADS));
    int bin_blks = (cross + sqrt_num_threads - 1 ) / sqrt_num_threads;
    for( int step = 0; step < NSTEPS; step++ )
    {
        //
        //  compute forces
        //


        compute_forces_gpu << blocks, NUM_THREADS >> (d_particles, gpu_bins, n_rows, n_cols, bin_side, n);
        //
        //  move particles
        //
        move_gpu << blocks, NUM_THREADS >> (d_particles, n, size);
        hipDeviceSynchronize();


        if (step % 4 == 0) {
            hipError_t error = hipMemcpy(particles, d_particles, n * sizeof(particle_t), hipMemcpyDeviceToHost);
            if (error != hipSuccess) {
                printf("1. %s \n", hipGetErrorString(error));
                exit(1);
            }
            for (int i = 0; i < cross; i++) {
                cpu_bins[i].length = 0;
            }

            for ( int i = 0; i < n; i++) {
                int x_ind = floor(particles[i].x / bin_side);
                int y_ind = floor(particles[i].y / bin_side);
                cpu_bins[x_ind*n_cols + y_ind].particles[cpu_bins[x_ind*n_cols + y_ind].length] = i;
                cpu_bins[x_ind*n_cols + y_ind].length += 1;
            }
            hipError_t error1 = hipMemcpy(gpu_bins, cpu_bins, cross * sizeof(sampleBin), hipMemcpyHostToDevice);
            if (error1 != hipSuccess) {
                printf("2. %s \n", hipGetErrorString(error1));
                exit(1);
            }
        }

        //
        //  save if necessary
        //
        if( fsave && (step%SAVEFREQ) == 0 ) {
            hipDeviceSynchronize();
            // Copy the particles back to the CPU
            hipMemcpy(particles, d_particles, n * sizeof(particle_t), hipMemcpyDeviceToHost);
            save( fsave, n, particles);
        }
    }
    hipDeviceSynchronize();
    simulation_time = read_timer( ) - simulation_time;

    printf( "CPU-GPU copy time = %g seconds\n", copy_time);
    printf( "n = %d, simulation time = %g seconds\n", n, simulation_time );

    free( particles );
    hipFree(d_particles);
    if( fsave )
        fclose( fsave );

    return 0;
}

